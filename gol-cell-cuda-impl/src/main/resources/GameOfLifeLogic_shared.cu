#include "hip/hip_runtime.h"
extern "C"
__global__ void calculate(int width, int height, int *board, int *board_result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int size = width * height;
    __shared__ int alive[size];
    if (i < size)
    {
        int cell_width = i % width;
        int cell_height = i / width;
        int cell_state = board[width*cell_height+cell_width];

        int alive_cells = 0;
        for(int x=-1; x<=1; x++){
            for(int y=-1; y<=1; y++){
                if(x==0 && y==0)
                    continue;
                int neighbour_cell = width*(cell_height+y)+cell_width+x;
                if(board[neighbour_cell] != 0)
                    alive_cells++;
            }
        }
        alive[i] = alive_cells;
        __syncthreads();


        if(cell_state != 0 && (alive_cells == 2 || alive_cells == 3)){
            board_result[i] = board[width*cell_height+cell_width];
        }

        else if(alive_cells == 3 && cell_state == 0){
            int red_count = 0;
            int green_count = 0;
            int blue_count = 0;
            int yellow_count = 0;
            int max_count = 0;
            int max_color = 0;

            for(int x=-1; x<=1; x++){
                for(int y=-1; y<=1; y++){
                    if(x==0 && y==0)
                        continue;
                    int neighbour_cell = width*(cell_height+y)+cell_width+x;
                    int color_neighbour_cell = board[neighbour_cell];
                    if(color_neighbour_cell == 2){
                        red_count++;
                        if(max_count < red_count){
                            max_count = red_count;
                            max_color = color_neighbour_cell;
                        }
                    }
                    else if(color_neighbour_cell == 3){
                        green_count++;
                        if(max_count < green_count){
                            max_count = green_count;
                            max_color = color_neighbour_cell;
                        }
                    }
                    else if(color_neighbour_cell == 4){
                        blue_count++;
                        if(max_count < blue_count){
                            max_count = blue_count;
                            max_color = color_neighbour_cell;
                        }
                    }
                    else if(color_neighbour_cell == 5){
                        yellow_count++;
                        if(max_count < yellow_count){
                            max_count = yellow_count;
                            max_color = color_neighbour_cell;
                        }
                    }

                }
            }
            int zero_color = 0;
            if(red_count == 0)
                zero_color = 2;
            if(green_count == 0)
                zero_color = 3;
            if(blue_count == 0)
                zero_color = 4;
            if(yellow_count == 0)
                zero_color = 5;

            if(max_count != 1)
                board_result[i] = max_color;
            else
                board_result[i] = zero_color;
        }
        else{
            board_result[i] = 0;
        }
    }

}
